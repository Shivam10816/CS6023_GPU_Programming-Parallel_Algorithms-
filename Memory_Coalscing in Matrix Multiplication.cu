#include<iostream>
#include<sys/time.h>
#include<hip/hip_runtime.h>
using namespace std;




__global__ void matrix_multiply_kernel(int* A, int* B, int* C, int* D, int* E, int p, int q, int r) {
    // Each thread computes one element of the output matrix E
    int i = blockIdx.x;
    int j = threadIdx.x;
    
    __shared__ int Ecol[1024];
    
    if(j==0){
        for(int k=0;k<p;k++){
            Ecol[k]=0;
        }
    }
    __syncthreads();
    
    for(int k=0;k<p;k++){
        int val = A[k*q+j]*B[j*r+i];
        atomicAdd(&Ecol[k],val);
    }
    
    for(int k=0;k<p;k++){
        int val = C[k*q+j]*D[i*q+j];
        atomicAdd(&Ecol[k],val);
    }
    
    __syncthreads();
    
    if(j==0){
       
       for(int k=0;k<p;k++){
            E[k*r+i]=Ecol[k];
        } 
    }
    
    
 
    
}


// function to compute the output matrix
void computE(int p, int q, int r, int *h_matrixA, int *h_matrixB, 
	         int *h_matrixC, int *h_matrixD, int *h_matrixE){
	// Device variables declarations...
	int *d_matrixA, *d_matrixB, *d_matrixC, *d_matrixD, *d_matrixE;
	
	// allocate memory...
	hipMalloc(&d_matrixA, p * q * sizeof(int));
	hipMalloc(&d_matrixB, q * r * sizeof(int));
	hipMalloc(&d_matrixC, p * q * sizeof(int));
	hipMalloc(&d_matrixD, r * q * sizeof(int));
	hipMalloc(&d_matrixE, p * r * sizeof(int));

	// copy the values...
	hipMemcpy(d_matrixA, h_matrixA, p * q * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixB, h_matrixB, q * r * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixC, h_matrixC, p * q * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixD, h_matrixD, r * q * sizeof(int), hipMemcpyHostToDevice);

	/* ****************************************************************** */
	/* Write your code here */
	/* Configure and launch kernels */
  

    matrix_multiply_kernel<<<r,q>>>(d_matrixA, d_matrixB, d_matrixC, d_matrixD, d_matrixE, p, q, r);

	/* ****************************************************************** */

	// copy the result back...
	hipMemcpy(h_matrixE, d_matrixE, p * r * sizeof(int), hipMemcpyDeviceToHost);

	// deallocate the memory...
	hipFree(d_matrixA);
	hipFree(d_matrixB);
	hipFree(d_matrixC);
	hipFree(d_matrixD);
	hipFree(d_matrixE);
}

// function to read the input matrices from the input file
void readMatrix(FILE *inputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fscanf(inputFilePtr, "%d", &matrix[i*cols+j]);
		}
	}
}

// function to write the output matrix into the output file
void writeMatrix(FILE *outputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fprintf(outputFilePtr, "%d ", matrix[i*cols+j]);
		}
		fprintf(outputFilePtr, "\n");
	}
}

int main(int argc, char **argv) {
	// variable declarations
	int p, q, r;
	int *matrixA, *matrixB, *matrixC, *matrixD, *matrixE;
	struct timeval t1, t2;
	double seconds, microSeconds;

	// get file names from command line
	char *inputFileName = argv[1];
	char *outputFileName = argv[2];

	// file pointers
	FILE *inputFilePtr, *outputFilePtr;
    
    inputFilePtr = fopen(inputFileName, "r");
	if(inputFilePtr == NULL) {
	    printf("Failed to open the input file.!!\n"); 
		return 0;
	}

	// read input values
	fscanf(inputFilePtr, "%d %d %d", &p, &q, &r);

	// allocate memory and read input matrices
	matrixA = (int*) malloc(p * q * sizeof(int));
	matrixB = (int*) malloc(q * r * sizeof(int));
	matrixC = (int*) malloc(p * q * sizeof(int));
	matrixD = (int*) malloc(r * q * sizeof(int));
	readMatrix(inputFilePtr, matrixA, p, q);
	readMatrix(inputFilePtr, matrixB, q, r);
	readMatrix(inputFilePtr, matrixC, p, q);
	readMatrix(inputFilePtr, matrixD, r, q);

	// allocate memory for output matrix
	matrixE = (int*) malloc(p * r * sizeof(int));

	// call the compute function
	gettimeofday(&t1, NULL);
	computE(p, q, r, matrixA, matrixB, matrixC, matrixD, matrixE);
	hipDeviceSynchronize();
	gettimeofday(&t2, NULL);

	// print the time taken by the compute function
	seconds = t2.tv_sec - t1.tv_sec;
	microSeconds = t2.tv_usec - t1.tv_usec;
	printf("Time taken (ms): %.3f\n", 1000*seconds + microSeconds/1000);

	// store the result into the output file
	outputFilePtr = fopen(outputFileName, "w");
	writeMatrix(outputFilePtr, matrixE, p, r);

	// close files
	fclose(inputFilePtr);
	fclose(outputFilePtr);

	// deallocate memory
	free(matrixA);
	free(matrixB);
	free(matrixC);
	free(matrixD);
	free(matrixE);

	return 0;
}
	
